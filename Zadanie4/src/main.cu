/*
 ============================================================================
 Name        : Zadanie3.cu
 Author      : Majkelo-Pęczkowiniki
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <random>
#include <ctime>
#include <string>
#include <sstream>
#include <cstdio>

#define _USE_MATH_DEFINES
#include <cmath>

#define G (6.67408*pow(10,-11))

//#define G 2

const double minStartedPositionValue = -10.0;
const double maxStartedPositionValue = 10.0;

const double maxStartedWeightValue = 10000.0;
const double maxStartedVelocityValue = 15.0;

const double timeDifference = 0.3;

using namespace std;

struct position {
	double x { };
	double y { };
	double z { };
};

struct physic_body {
	position r { };
	double v { };
	double m { };
};

__global__ void universeUpdateVelocity(int* N, physic_body* tab) {
	int index;
	index = 2 * (threadIdx.x + blockDim.x * blockIdx.x);

	if (index < (*N)) {
		double solution = 0;
		for (int j = 0; j < (*N); ++j) {
			if (j == index) {
				continue;
			}

			double dist = sqrt(
					pow(tab[index].r.x - tab[j].r.x, 2)
							+ pow(tab[index].r.y - tab[j].r.y, 2)
							+ pow(tab[index].r.z - tab[j].r.z, 2));

			solution += (tab[j].m / (pow(dist, 3))) * dist;
		}
		printf("Body %d velocity: %lf \n", index, tab[index].v);
		printf("Body %d velocity changing by: %lf \n", index,
				tab[index].v + G * solution * timeDifference);
		tab[index].v = tab[index].v + G * solution * timeDifference;
		printf("Body %d velocity changed to %lf \n", index, tab[index].v);

	}
}

__global__ void universeUpdatePosition(int* N, physic_body* tab) {
	int index;
	index = 2 * (threadIdx.x + blockDim.x * blockIdx.x);

	if (index < (*N)) {
		tab[index].r.x = tab[index].r.x + tab[index].v * timeDifference;
		tab[index].r.y = tab[index].r.y + tab[index].v * timeDifference;
		tab[index].r.z = tab[index].r.z + tab[index].v * timeDifference;
		printf("Body %d position changed to: %lf %lf %lf \n", index, tab[index].r.x,
				tab[index].r.y, tab[index].r.z);
	}
}

timespec diff(timespec start, timespec end);
double computeDistance(position positionA, position positionB);
void generateBodyValues(physic_body* bodies, int N);
void computeOnCPU(physic_body* bodies, int N, int iterations);

int main(int argc, char* argv[]) {
	srand(time(NULL));

	string s(argv[1]);
	string s2(argv[2]);

	const int iterations = stof(s2);

	const int N = stoi(s);

	physic_body cpuBodies[N];
	physic_body gpuBodies[N];
	generateBodyValues(cpuBodies, N);
	generateBodyValues(gpuBodies, N);

	cout << "G" << G << endl;

//	computeOnCPU(cpuBodies, N, iterations);

//NOW CUDA

//Ogarniane jest tutaj ile dana karta może wytrzymać wątków na jeden blok
	int minGridSize;
	int blockSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			(void*) universeUpdateVelocity, 0, N / 2);

	int gridSize = (N / 2 + blockSize - 1) / blockSize;

	dim3 thread(blockSize);
	dim3 block(gridSize);

	size_t bodySize = sizeof(physic_body);
	size_t sizeOfTab = N * bodySize;

	//Przedrostek d_ oznacza, ¿e dana zmienna wykorzystywana jest nie na procku, ale na karcie
	physic_body* d_tab;
	int* d_N;

	hipMalloc(&d_tab, sizeOfTab);
	hipMemcpy(d_tab, &gpuBodies, sizeOfTab, hipMemcpyHostToDevice);

	hipMalloc(&d_N, sizeof(int));
	hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);

	for (int iteration = 0; iteration < iterations; ++iteration) {
		universeUpdateVelocity<<< block, thread >>>(d_N, d_tab);
		universeUpdatePosition<<< block, thread >>>(d_N, d_tab);
	}

	hipFree(d_tab);
	hipFree(d_N);

	printf("End of all \n");

	return 0;
}

void generateBodyValues(physic_body* bodies, int N) {
	for (int i = 0; i < N; ++i) {
		bodies[i].r.x = minStartedPositionValue
				+ (rand()
						% (int) (maxStartedPositionValue
								- minStartedPositionValue + 1));
		bodies[i].r.y = minStartedPositionValue
				+ (rand()
						% (int) (maxStartedPositionValue
								- minStartedPositionValue + 1));
		bodies[i].r.z = minStartedPositionValue
				+ (rand()
						% (int) (maxStartedPositionValue
								- minStartedPositionValue + 1));
		bodies[i].v = (rand() % (int) (maxStartedVelocityValue + 1));
		bodies[i].m = (rand() % (int) (maxStartedWeightValue + 1));
	}
}

timespec diff(timespec start, timespec end) {
	timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0) {
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	return temp;
}

double computeDistance(position positionA, position positionB) {
	return sqrt(
			pow(positionA.x - positionB.x, 2)
					+ pow(positionA.y - positionB.y, 2)
					+ pow(positionA.z - positionB.z, 2));
}

void computeOnCPU(physic_body* bodies, int N, int iterations) {
	for (int iter = 0; iter < iterations; ++iter) {
		printf("Iteration %d \n", iter + 1);
		//Change velocity
//		printf ("Updating velocity\n");
		for (int i = 0; i < N; ++i) {
			double solution = 0;
			for (int j = 0; j < N; ++j) {
				if (j == i) {
					continue;
				}
				double dist = computeDistance(bodies[i].r, bodies[j].r);
				solution += (bodies[j].m / (pow(dist, 3))) * dist;
			}
//			printf ("Body %d velocity: %lf \n", i, bodies[i].v);
//			printf ("Body %d velocity changing by: %lf \n", i, bodies[i].v + G * solution * timeDifference);
			bodies[i].v = bodies[i].v + G * solution * timeDifference;
			printf("Body %d velocity changed to %lf \n", i, bodies[i].v);
		}

		//Change position
//		printf ("Updating position\n");
		for (int i = 0; i < N; ++i) {
//			printf ("Body %d position: %lf %lf %lf \n", i, bodies[i].r.x, bodies[i].r.y, bodies[i].r.z);
			bodies[i].r.x = bodies[i].r.x + bodies[i].v * timeDifference;
			bodies[i].r.y = bodies[i].r.y + bodies[i].v * timeDifference;
			bodies[i].r.z = bodies[i].r.z + bodies[i].v * timeDifference;
			printf("Body %d position changed to: %lf %lf %lf \n", i,
					bodies[i].r.x, bodies[i].r.y, bodies[i].r.z);
		}
	}
}
