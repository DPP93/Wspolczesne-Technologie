#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <ctime>
#include <string>

using namespace std;

inline hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess) {
		cout << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
	}
	return result;
}

__device__ float calcForce(float massJ, float distance, float g_const,
		float b_const) {
	if (distance < 0.01) {
        distance = 0.01;
	}
    return b_const * (g_const * (massJ) / ((distance * distance * distance)));
}

__global__ void KernelForces(unsigned int n, float deltaT, float* m, float3 *p,
		float3 *v, float3 *f) {
	int bd = blockDim.x * blockIdx.x + threadIdx.x;
	int numThreads = blockDim.x * gridDim.x;

	float G = 6.672 * 10e-11;
	float SOFTENING = 10e7;

	//compare all with all
	if (bd < n) {
		for (unsigned int ia = bd; ia < n; ia += numThreads) {
			float lfx = 0.0f;
			float lfy = 0.0f;
			float lfz = 0.0f;

			for (unsigned int ib = 0; ib < n; ib++) {
				//compute distance
				float dx = (p[ib].x - p[ia].x);
				float dy = (p[ib].y - p[ia].y);
				float dz = (p[ib].z - p[ia].z);

				float distance = sqrt((dx * dx) + (dy * dy) + (dz * dz));

				//compute force
				lfx += calcForce(m[ib], distance, G, SOFTENING) * dx;
				lfy += calcForce(m[ib], distance, G, SOFTENING) * dy;
				lfz += calcForce(m[ib], distance, G, SOFTENING) * dz;
			}

			f[ia] = make_float3(lfx, lfy, lfz);

			v[ia].x += deltaT * f[ia].x;
			v[ia].y += deltaT * f[ia].y;
			v[ia].z += deltaT * f[ia].z;
		}
	}
}

__global__ void KernelPositions(unsigned int n, float deltaT, float3 *p,
		float3 *v) {
	int bd = blockDim.x * blockIdx.x + threadIdx.x;
	int numThreads = blockDim.x * gridDim.x;

	//compare all with all
	if (bd < n) {
		for (unsigned int ia = bd; ia < n; ia += numThreads) {

			p[ia].x += deltaT * v[ia].x;
			p[ia].y += deltaT * v[ia].y;
			p[ia].z += deltaT * v[ia].z;
		}
	}
}

float randomFloat(float a, float b) {
	float random = ((float) rand()) / (float) RAND_MAX;
	float diff = b - a;
	float r = random * diff;
	return a + r;
}

int main(int argc, char* argv[]) {
	string s(argv[1]);
	string s2(argv[2]);
	string s3(argv[3]);
	string s4(argv[4]);
	string s5(argv[5]);
	string s6(argv[6]);
	string s7(argv[7]);
	string s8(argv[8]);
	const int n = stoi(s), iterations = stoi(s2), inHowMuchIterationsPrint = stoi(s4);
	float deltaT = stof(s3);
	float massMin = stof(s5), massMax = stof(s6), initP = stof(s7), initV = stof(s8);

	float m[n];
	float3 p[n], v[n], f[n];

	/*
	 * m - masa
	 * p - wektor pozycji
	 * v - wektor prędkości
	 * f - wektor sił
	 * */

	float *_m;
	float3 *_p, *_v, *_f;

	srand(time(NULL));

	for (int i = 0; i < n; ++i) {
		m[i] = randomFloat(massMin, massMax);
		p[i] = make_float3(randomFloat(-initP, initP), 
                           randomFloat(-initP, initP),
                           randomFloat(-initP, initP));
		//v[i] = make_float3(randomFloat(-2, 2), randomFloat(-2, 2), randomFloat(-2, 2));
		v[i] = make_float3(randomFloat(-initV, initV),
                           randomFloat(-initV, initV),
                           randomFloat(-initV, initV));
		f[i] = make_float3(randomFloat(0, 0), randomFloat(0, 0), randomFloat(0, 0));
	}

	int floatSize = sizeof(float);
	int float3Size = sizeof(float3);
	int floatVectorSize = n * floatSize;
	int float3VectorSize = n * float3Size;

	checkCuda(hipMalloc(&_m, floatVectorSize));
	checkCuda(hipMalloc(&_p, float3VectorSize));
	checkCuda(hipMalloc(&_v, float3VectorSize));
	checkCuda(hipMalloc(&_f, float3VectorSize));

	checkCuda(hipMemcpy(_m, m, floatVectorSize, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(_p, p, float3VectorSize, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(_v, v, float3VectorSize, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(_f, f, float3VectorSize, hipMemcpyHostToDevice));

	int minGridSize;
	int blockSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			(void*) KernelForces, 0, n);

	int gridSize = (n + blockSize - 1) / blockSize;

//Tutaj printuję dane z bloków i wątków coby pokazać jak one wyglądają
	dim3 thread(blockSize);
	dim3 block(gridSize);

	cout << "Block: " << block.x << " " << block.y << " " << block.z << endl;
	cout << "Thread: " << thread.x << " " << thread.y << " " << thread.z
			<< endl;

	for (int k = 0; k < n; k++) {
        printf("Body %d mass: %lf\n", k, m[k]);
    }

	printf("Time: %lf s\n", 0.0);
	for (int k = 0; k < n; k++) {
		printf("Body %d position: %lf %lf %lf\n", k, p[k].x, p[k].y, p[k].z);
	}

	for (int i = 1; i <= iterations; ++i) {
//		printf("Iteration %d\n", i);
		KernelForces<<<block , thread>>> (n, deltaT, _m, _p, _v, _f);
		KernelPositions<<<block , thread>>> (n, deltaT, _p, _v);
		if (i % inHowMuchIterationsPrint == 0) {
			printf ("Time: %lf s\n", deltaT*i);
			checkCuda(hipMemcpy(p, _p, float3VectorSize, hipMemcpyDeviceToHost));
			for (int k = 0; k < n; k++) {
				printf("Body %d position: %lf %lf %lf\n", k, p[k].x, p[k].y, p[k].z);
			}
		}
	}

	hipDeviceSynchronize();

	hipFree(_m);
	hipFree(_p);
	hipFree(_v);
	hipFree(_f);
	hipDeviceReset();
	hipDeviceReset();
	cout << "END" << endl;
}
