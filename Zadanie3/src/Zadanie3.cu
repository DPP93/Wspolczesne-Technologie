/*
 ============================================================================
 Name        : Zadanie3.cu
 Author      : Majkelo-Pęczkowiniki
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */



#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <random>
#include <ctime>
#include <string>
#include <sstream>
#include <cstdio>

using namespace std;

__global__ void bubble(int* N, int* tab)
{
	int ind, i, j, s;
	int a, b;
	ind = 2 * (threadIdx.x + blockDim.x*blockIdx.x);
	printf("%d\n", ind);
	for (int k = 0; k<(*N) - 1; k++)
	{

		s = (k % 2);
		i = ind + s;
		j = ind + 1 + s;
		if (j < (*N))
		{
			a = tab[i];
			b = tab[j];
			if (b<a)
			{
				tab[i] = b;
				tab[j] = a;
			}
		}
		__syncthreads();
	}
}

__global__ void bubble1(int* N, int* s, int* tab) {
	int ind, i, j;
	int a, b;
	ind = 2 * (threadIdx.x + blockDim.x*blockIdx.x);
	i = ind + (*s);
	j = ind + (*s) + 1;

	if (j < (*N)) {
		a = tab[i];
		b = tab[j];

		if (b < a) {
			tab[i] = b;
			tab[j] = a;
		}
	}
}

timespec diff(timespec start, timespec end);
void generateRandomData(int* tab, int N, int min, int max);
void computeBubbleOnCPU(int* tab, int N);

int main(int argc, char* argv[])
{
	srand(time(NULL));

	string s(argv[1]);
	string s2(argv[2]);
	string s3(argv[3]);

	int min, max;

	min = stof(s2);
	max = stof(s3);

	const int N = stoi(s);
	int tabBase[N]; //= new int[N];
	int tabCPU[N]; //= new int[N];
	int tabGPU[N]; //= new int[N];

	generateRandomData(tabBase, N, min, max);

	for (int i = 0; i < N; ++i) {
		tabCPU[i] = tabBase[i];
		tabGPU[i] = tabBase[i];
	}

	timespec start, stop;
	timespec returnTime;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);

	computeBubbleOnCPU(tabCPU, N);


	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	returnTime = diff(start, stop);
	cout << "CPU time: " << returnTime.tv_sec << "."
         << setfill('0') << setw(9) << returnTime.tv_nsec << "s" << endl;

	//Ogarniane jest tutaj ile dana karta może wytrzymać wątków na jeden blok
    int minGridSize;
    int blockSize;
    hipOccupancyMaxPotentialBlockSize(
            &minGridSize, &blockSize, (void*)bubble, 0, N
    );

    int blockHeight = blockSize;
    int blockWidth = 1;
    while (blockHeight > blockWidth) {
        blockHeight /= 2;
        blockWidth *= 2;
    }

    int gridWidth = (N + blockWidth - 1) / blockWidth;

	dim3 thread (blockWidth);
	dim3 block (gridWidth);

	//Poalokowane coby sobie nie robiæ problemów przy alokacji rozmiarów na karcie graficznej
	size_t intSize = sizeof(int);
	size_t sizeOfTab = N * intSize;

	//Przedrostek d_ oznacza, ¿e dana zmienna wykorzystywana jest nie na procku, ale na karcie
	int* d_tab;
	int* d_N;
	int* d_s;

	//Zaalokuj pamiêæ na karcie na wektor do mno¿enia
	hipMalloc(&d_tab, sizeOfTab);
	hipMemcpy(d_tab, &tabGPU, sizeOfTab, hipMemcpyHostToDevice);

	hipMalloc(&d_N, sizeof(int));
	hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_s, sizeof(int));

	//Tutaj printujê dane z bloków i w¹tków coby pokazaæ jak one wygl¹daj¹
	cout << "Block: " << block.x << " " << block.y << " " << block.z << endl;
	cout << "Thread: " << thread.x << " " << thread.y << " " << thread.z << endl;

//	bubble <<< block, thread>>> (d_N, d_tab);


	hipEvent_t cudaStart, cudaStop;
	hipEventCreate(&cudaStart);
	hipEventCreate(&cudaStop);

	float milliseconds = 0, temp = 0;


	for (int i = 0; i < N - 1; ++i) {
		int k = (i%2);
		hipMemcpy(d_s, &k, sizeof(int), hipMemcpyHostToDevice);
		hipEventRecord(cudaStart);
		bubble1 <<< block, thread >>>(d_N, d_s, d_tab);
		hipEventRecord(cudaStop);
		hipEventSynchronize(cudaStop);
		hipEventElapsedTime(&temp, cudaStart, cudaStop);
		milliseconds += temp;
	}


	hipEventSynchronize(cudaStop);
	//Poczekaæ na zakoñczenie wszystkiego
	hipDeviceSynchronize();

	cout << "GPU time: " << milliseconds << "ms" << endl;

	hipMemcpy(tabGPU, d_tab, sizeOfTab, hipMemcpyDeviceToHost);

	hipFree(d_tab);
	hipFree(d_N);
	hipFree(d_s);

	hipDeviceReset();

//	for (int i : tabCPU) {
//		std::cout << i << " ";
//	}
//	std::cout << std::endl;
//
//	for (int i : tabGPU) {
//		std::cout << i << " ";
//	}
//	std::cout << std::endl;

//	delete[] tabBase;
//	delete[] tabCPU;
//	delete[] tabGPU;

	for (int i = 0; i < N; ++i) {

//		cout << (i+1) << " - " << tabCPU[i] << "   :   " << tabGPU[i] << endl;
		if (tabCPU[i] != tabGPU[i]) {
			cout << "What a Terrible Failure" << endl;
			break;
		}
	}

//	getchar();
	return 0;
}

void generateRandomData(int* tab, int N, int min, int max) {
	for (int x = 0; x < N; ++x) {
		tab[x] = (min + (rand() % (int)(max - min + 1)));
	}
}

void computeBubbleOnCPU (int* tab, int N) {
	int counter  = N;

	do
	{
		for (int i = 0; i < counter - 1; ++i) {
			if (tab[i] > tab[i+1]) {
				int temp = tab[i];
				tab[i] = tab[i+1];
				tab[i+1] = temp;
			}
		}
		--counter;
	}while (counter > 1);

}

timespec diff(timespec start, timespec end) {
	timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0) {
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	return temp;
}
